#include "pybind11/pybind11.h"
#include "pybind11/iostream.h"
#include "pybind11/complex.h"
#include "pybind11/stl.h"
#include "pybind11/numpy.h"

#include <hip/hip_runtime.h>

#include "parser.h"
#include "fileIO.h"

#define PYBIND11_EXPORT __attribute__ ((visibility("default")))

namespace py = pybind11;
using namespace FileIO;

void test_all(){
    printf("Please use the binary 'gpue' for unit testing the C++/CUDA modules");    
}

void fileio_binding(py::module &m){
    m.def("init", &init);
    m.def("load", &load);
    m.def("loadA", &loadA);
    m.def("writeOutEnergy", &writeOutEnergy);
    m.def("writeOutWfc", &writeOutWfc);
    m.def("writeOutV", &writeOutV);
    m.def("writeOutEdges", &writeOutEdges);
    m.def("writeOutAx", &writeOutAx);
    m.def("writeOutAy", &writeOutAy);
    m.def("writeOutAz", &writeOutAz);
    m.def("writeOutX", &writeOutX);
    m.def("writeOutY", &writeOutY);
    m.def("writeOutZ", &writeOutZ);
    m.def("writeOutParams", &writeOutParams);
    m.def("destroy", &destroy);
    m.def("writeOutInt", &writeOutInt);
    m.def("writeOutVortex", &writeOutVortex);
    m.def("writeOutAdjMat", py::overload_cast<std::string, int*, unsigned int*, int, int>(&writeOutAdjMat), "");
    m.def("writeOutAdjMat", py::overload_cast<std::string, double*, unsigned int*, int, int>(&writeOutAdjMat), "");
}

PYBIND11_MODULE(_PyGPUE_IO, m){
    fileio_binding(m);
    m.def("test_all", &test_all);
    //parser.h
    m.def("parseArgs", 
        [](std::vector<std::string> args) {
            std::vector<char *> cstrs;
            cstrs.reserve(args.size());
            for (auto &s : args) {
                cstrs.push_back(const_cast<char *>(s.c_str()));
            }
            return parseArgs(cstrs.size(), cstrs.data());
        }
    );
}


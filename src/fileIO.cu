#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "../include/fileIO.h"

namespace FileIO{

    /*
     * Reads datafile into memory.
     */
    double2* readIn(std::string fileR, std::string fileI,
                        int gSize){
        FILE *f;
        f = fopen(fileR.c_str(),"r");
        int i = 0;
        double2 *arr = (double2*) malloc(sizeof(double2)*gSize);
        double line;
        while(fscanf(f,"%lE",&line) > 0){
            arr[i].x = line;
            ++i;
        }
        fclose(f);
        f = fopen(fileI.c_str(),"r");
        i = 0;
        while(fscanf(f,"%lE",&line) > 0){
            arr[i].y = line;
            ++i;
        }
        fclose(f);
        return arr;
    }

    /*
     * Writes out the parameter file.
     */
    void writeOutParam(Grid &par, std::string file){
        par.write(file);
    }

    /*
     * Writes out double2 complex data files.
     */
    void writeOut(std::string buffer, std::string file, double2 *data,
                      int length, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        int i;
        for (i = 0; i < length; i++)
            fprintf (f, "%.16e\n",data[i].x);
        fclose (f);

        sprintf ((char *)buffer.c_str(), "%si_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        for (i = 0; i < length; i++)
            fprintf (f, "%.16e\n",data[i].y);
        fclose (f);

    }

    /*
     * Writes out double type data files.
     */
    void writeOutDouble(std::string file, double *data, int length, int step){
        std::ofstream output;
        output.open(file + "_" + std::to_string(step));
        for (int i = 0; i < length; ++i){
            output << data[i] << '\n';
        }

        output.close();
    }

    /*
     * Writes out bool type data files.
     */
    void writeOutBool(std::string file, bool *data,int length, int step){
        std::ofstream output;
        output.open(file + "_" + std::to_string(step));
        for (int i = 0; i < length; ++i){
            output << data[i] << '\n';
        }

        output.close();
    }

    /*
     * Writes out int type data files.
     */
    void writeOutInt(std::string buffer, std::string file, int *data,
                         int length, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        int i;
        for (i = 0; i < length; i++)
            fprintf (f, "%d\n",data[i]);
        fclose (f);
    }

    /*
     * Writes out int2 data type.
     */
    void writeOutInt2(std::string buffer, std::string file, int2 *data,
                          int length, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        int i;
        for (i = 0; i < length; i++)
            fprintf (f, "%d,%d\n",data[i].x,data[i].y);
        fclose (f);
    }

    /*
     * Writes out tracked vortex data.
     */
    void writeOutVortex(std::string buffer, std::string file,
                            std::vector<std::shared_ptr<Vtx::Vortex>> &data, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);

        f = fopen (buffer.c_str(),"w");
        int i;

        fprintf (f, "#UID,X,Xd,Y,Yd,WINDING,isOn\n");
        for (i = 0; i < data.size(); i++)
            //fprintf (f, "%d,%d,%e,%d,%e,%d\n",data[i]->getUID(),data[i]->getCoords().x,data[i]->getCoordsD().x,data[i]->getCoords().y,data[i]->getCoordsD().y,data[i]->getWinding());
            fprintf (f, "%d,%e,%d,%e,%d\n",data[i]->getCoords().x,data[i]->getCoordsD().x,data[i]->getCoords().y,data[i]->getCoordsD().y,data[i]->getWinding());
        fclose (f);
    }

    /*
     * Opens and closes file. Nothing more. Nothing less.
     */
    int readState(std::string name){
        FILE *f;
        f = fopen(name.c_str(),"r");
        fclose(f);
        return 0;
    }

    /*
     * Outputs the adjacency matrix to a file
     */
    void writeOutAdjMat(std::string buffer, std::string file, int *mat, unsigned int *uids, int dim, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        fprintf (f, "(*");
        for(int ii = 0; ii<dim; ++ii){
            fprintf (f, "%d",uids[ii]);
        }
        fprintf (f, "*)\n");
        fprintf (f, "{\n");
        for(int ii = 0; ii < dim; ++ii){
            fprintf (f, "{");
            for(int jj = 0; jj < dim; ++jj){
                fprintf (f, "%i",mat[ii*dim + jj]);
                if(jj<dim-1)
                    fprintf (f, ",");
                else
                    fprintf (f, "}");
            }
            if(ii<dim-1)
                fprintf (f, ",");
            fprintf (f, "\n");
        }
        fprintf (f, "}\n");
        fclose(f);
    }
    void writeOutAdjMat(std::string buffer, std::string file, double *mat,
                        unsigned int *uids, int dim, int step){
        FILE *f;
        sprintf ((char *)buffer.c_str(), "%s_%d", file.c_str(), step);
        f = fopen (buffer.c_str(),"w");
        fprintf (f, "(*");
        for(int ii = 0; ii<dim; ++ii){
            fprintf (f, "%d",uids[ii]);
            if(ii!=dim-1)
               /* I am not sure what Lee wants here, but I think...
                           fprintf (f, ",",uids[ii]); */
                           fprintf (f, ",");

        }
        fprintf (f, "*)\n");
        fprintf (f, "{\n");
        for(int ii = 0; ii < dim; ++ii){
            fprintf (f, "{");
            for(int jj = 0; jj < dim; ++jj){
                fprintf (f, "%e",mat[ii*dim + jj]);
                if(jj<dim-1)
                    fprintf (f, ",");
                else
                    fprintf (f, "}");
            }
            if(ii<dim-1)
                fprintf (f, ",");
            fprintf (f, "\n");
        }
        fprintf (f, "}\n");
        fclose(f);
    }
}

#include "init.h"
#include "dynamic.h"
#include "split_op.h"

int main(int argc, char **argv){

    Grid par = parseArgs(argc,argv);

    int device = par.ival("device");
    int dimnum = par.ival("dimnum");
    int wfc_num = par.ival("wfc_num");
    cudaHandleError(hipSetDevice(device));

    time_t start,fin;
    time(&start);
    printf("Start: %s\n", ctime(&start));

    //************************************************************//
    /*
    * Initialise the Params data structure to track params and variables
    */
    //************************************************************//

    // If we want to read in a wfc, we may also need to imprint a phase. This
    // will be done in the init_2d and init_3d functions
    // We need a number of parameters for now
    int xDim = par.ival("xDim");
    int yDim = par.ival("yDim");
    int zDim = par.ival("zDim");
    if(par.bval("read_file") == true){
        FileIO::load(par);
    }

    if(par.bval("corotating_override")){
        std::cout << "Overriding rotational flag set by file\n";
        par.store("corotating", false);
    }

    init(par);

    int gsteps = par.ival("gsteps");
    int esteps = par.ival("esteps");
    std::string data_dir = par.sval("data_dir");
    std::cout << "variables re-established" << '\n';

    if (par.bval("write_file")){
        FileIO::writeOutParams(par);
    }

    if(gsteps > 0){
        std::cout << "Imaginary-time evolution started..." << '\n';
        par.store("gstate", true);
        set_variables(par);

        evolve(par, gsteps);
    }

    if(esteps > 0){
        std::cout << "real-time evolution started..." << '\n';
        par.store("gstate", false);
        set_variables(par);

        evolve(par, esteps);
    }

    par.store("found_sobel",false);
    FileIO::writeOutParams(par);

    // Close the output file
    FileIO::destroy();

    std::cout << "done evolving" << '\n';
    time(&fin);
    printf("Finish: %s\n", ctime(&fin));
    printf("Total time: %ld seconds\n ",(long)fin-start);
    std::cout << '\n';
    return 0;
}

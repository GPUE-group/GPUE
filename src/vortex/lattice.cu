#include "hip/hip_runtime.h"

#include "lattice.h"
#include <iostream>

using namespace LatticeGraph;

// Constructor and destructor

Lattice::Lattice(){
}

Lattice::~Lattice(){
	this->getVortices().clear();
	this->getEdges().clear();
}

// Getters

std::vector< std::shared_ptr<Node> >& Lattice::getVortices(){
	return this->vortices;
}

std::shared_ptr<Node> Lattice::getVortexIdx(unsigned int idx){
	return getVortices().at(idx);
}

/***
 * Gets the location of the Node with UID uid.
 */
unsigned int Lattice::getVortexIdxUid(unsigned int uid){
	for (size_t ii=0; ii< getVortices().size(); ++ii){
		if(this->Lattice::getVortexIdx(ii)->getUid()== uid){
			return ii;
		}
	}
	return -1;
}

/***
 * Gets the the Node with UID uid. Assumes Node exists.
 */
std::shared_ptr<Node> Lattice::getVortexUid(unsigned int uid){
	for (std::shared_ptr<Node> n : this->Lattice::getVortices()){
		if(n->getUid()== uid){
			return n;
		}
	}
	return std::shared_ptr<Node>();
}

double Lattice::getVortexDistance(std::shared_ptr<Node> n1, std::shared_ptr<Node> n2){
	return sqrt(pow(n1->getData().getCoords().x - n2->getData().getCoords().x,2)
	            +  pow(n1->getData().getCoords().y - n2->getData().getCoords().y,2));
}

double Lattice::getVortexDistanceD(std::shared_ptr<Node> n1, std::shared_ptr<Node> n2){
	return sqrt(pow(n1->getData().getCoordsD().x - n2->getData().getCoordsD().x,2)
	            +  pow(n1->getData().getCoordsD().y - n2->getData().getCoordsD().y,2));
}

std::shared_ptr<Edge> Lattice::getEdgeIdx(unsigned int idx){
	return getEdges().at(idx);
}

/***
 * Gets the location of the Edge with UID uid.
 */
unsigned int Lattice::getEdgeIdxUid(unsigned int uid){
	for (size_t ii=0; ii< getEdges().size(); ++ii){
		if(this->Lattice::getEdgeIdx(ii)->getUid()== uid){
			return ii;
		}
	}
	return -1;
}

/***
 * Gets the the Edge with UID uid. Assumes Node exists.
 */
std::shared_ptr<Edge> Lattice::getEdgeUid(unsigned int uid){
	for (std::shared_ptr<Edge> e : this->Lattice::getEdges()){
		if(e->getUid()== uid){
			return e;
		}
	}
	return NULL;
}

std::vector< std::shared_ptr<Edge> >& Lattice::getEdges(){
	return this->edges;
}

// Setters

void Lattice::setVortex(unsigned int idx, std::shared_ptr<Node> n){
	this->Lattice::getVortices().at(idx)=(n);
}

void Lattice::setEdge(unsigned int idx, std::shared_ptr<Edge> e){
	this->Lattice::getEdges().at(idx)=(e);
}

// Creation


void Lattice::createEdges(unsigned int radius){
	std::shared_ptr<Edge> e;
	double dist = 0.0;
	for(size_t ii=0; ii< this->Lattice::getVortices().size(); ++ii){
		//std::cout << "Got here ii " << ii << std::endl;
		for(size_t jj=ii+1; jj < this->Lattice::getVortices().size(); ++jj){
			dist = Lattice::getVortexDistance(this->getVortexIdx(ii),this->getVortexIdx(jj));
			if(dist < radius ) {
				//std::cout << "Got here jj " << jj << std::endl;
				e.reset(new Edge ( this->getVortexIdx(ii), this->getVortexIdx(jj) ));
				e->setWeight(dist);
				this->Lattice::addEdge(e,this->getVortexIdx(ii),this->getVortexIdx(jj));
			}
		}
	}
}
void Lattice::createEdges(double radius){
	std::shared_ptr<Edge> e;
	double dist = 0.0;
	for(size_t ii=0; ii< this->Lattice::getVortices().size(); ++ii){
		//std::cout << "Got here ii " << ii << std::endl;
		for(size_t jj=ii+1; jj < this->Lattice::getVortices().size(); ++jj){
			dist = Lattice::getVortexDistance(this->getVortexIdx(ii),this->getVortexIdx(jj));
			if( dist < radius ) {
				//std::cout << "Got here jj " << jj << std::endl;
				e.reset(new Edge ( this->getVortexIdx(ii), this->getVortexIdx(jj) ));
				e->setWeight(dist);
				this->Lattice::addEdge(e,this->getVortexIdx(ii),this->getVortexIdx(jj));
			}
		}
	}
}

void Lattice::addVortex(std::shared_ptr<Node> n){
	this->Lattice::getVortices().push_back((n));
}

void Lattice::addEdge(std::shared_ptr<Edge> e){
	this->addEdge(e, e->getVortex(0).lock(), e->getVortex(1).lock());
}

void Lattice::addEdge(std::shared_ptr<Edge> e, std::shared_ptr<Node> n1, std::shared_ptr<Node> n2){
	this->Lattice::getEdges().push_back(e);
	std::weak_ptr<Edge> e1 = e;
	std::weak_ptr<Edge> e2 = e;
	n1->addEdge(e1);
	n2->addEdge(e2);
}

// Deletion

void Lattice::removeVortex(std::shared_ptr<Node> n){
	for(std::weak_ptr<Edge> e : n->getEdges()){
		if(e.lock()){
			std::cout << "UID: Removing Vortex{" << n->getUid() <<"}." << std::endl;
			this->removeEdge(e.lock());
			this->Lattice::getVortices().erase(this->Lattice::getVortices().begin() + this->getVortexIdxUid(n->getUid()));
		}
		else{
			std::cout << "Cannot remove UID:Edge{"<< e.lock()->getUid() << "}, does not exist." << std::endl;
		}
	}
}

void Lattice::removeVortexUid(unsigned int uid){
	auto vtx = this->getVortexUid(uid);
	if(vtx){
		this->Lattice::removeVortex(vtx);
	}
	else{
		std::cout << "Cannot remove UID:Vortex{"<< uid << "}, does not exist." << std::endl;
	}
}

void Lattice::removeVortexIdx(unsigned int idx){
	auto vtx = this->getVortexIdx(idx);
	if(vtx){
		this->Lattice::removeVortex(vtx);
	}
	else{
		std::cout << "Cannot remove IDX:Vortex["<< idx << "], does not exist." << std::endl;
	}
}

void Lattice::removeEdge(std::shared_ptr<Edge> e){
	std::cout << "Removing Edge{" << e->getUid() <<"} connecting Node{" << e->getVortex(0).lock()->getUid() << "} and Node{" << e->getVortex(1).lock()->getUid() << "}." << std::endl;
	e->getVortex(0).lock()->removeEdgeUid(e->getUid());
	e->getVortex(1).lock()->removeEdgeUid(e->getUid());
	this->Lattice::getEdges().erase(this->Lattice::getEdges().begin() + this->Lattice::getEdgeIdxUid(e->getUid()));
}

void Lattice::removeEdgeIdx(unsigned int idx){
	std::weak_ptr<Edge> e = this->getEdgeIdx(idx);
	if (auto el = e.lock()) {
		this->Lattice::removeEdge(el);
	}
	else{
		std::cout << "Cannot remove IDX:Edge[" << idx << "], does not exist." << std::endl;
	}
}

void Lattice::removeEdgeUid(unsigned int uid) {
	std::weak_ptr<Edge> e = this->getEdgeUid(uid);
	if (auto el = e.lock()) {
		this->Lattice::removeEdge(el);
	}
	else{
		std::cout << "Cannot remove UID:Edge{" << uid << "}, does not exist." << std::endl;
	}
}

void Lattice::removeEdge(std::shared_ptr<Node> n1, std::shared_ptr<Node> n2){
	std::weak_ptr<Edge> e = this->Lattice::isConnected(n1,n2);
	if(e.lock()){
		this->Lattice::removeEdge(e.lock());
	}
	else{
		std::cout << "Node{" << n1->getUid() << "} and Node{" << n2->getUid() << "} were unconnected." << std::endl;
	}

}

void Lattice::removeEdges(std::shared_ptr<Node> n1){
	//n1->removeEdges();
}


void Lattice::createVortex(double posx, double posy, int winding){

}

void Lattice::destroyVortex(unsigned int uid){
	this->Lattice::getVortexUid(uid);
}

// Generating matrices

/**
 * Create adjacency matrix
 */
void Lattice::genAdjMat(unsigned int *mat){
	int idx1, idx2, idx;
	idx1 = 0; idx2 = 0; idx=0;
	for(std::shared_ptr<Node> n : this->Lattice::getVortices()){
		idx1=this->getVortexIdxUid(n->getUid());
		for(std::weak_ptr<Edge> e : n->getEdges()){
			idx2 = this->getVortexIdxUid(n->getConnectedNode(e.lock())->getUid());
			//std::cout << "this=" << n->getUid() << "   connected=" << n->getConnectedNode(e.lock())->getUid() << std::endl;
			idx = idx1*this->Lattice::getVortices().size() + idx2;
			//std::cout << "idx1=" << idx1 << "   idx2=" << idx2 << " idx=" << idx << "\n" << std::endl;
			mat[idx] = 1;
		}
	}
}

void Lattice::genAdjMat(double *mat){
	int idx1, idx2, idx;
	idx1 = 0; idx2 = 0; idx=0;
	for(std::shared_ptr<Node> n : this->Lattice::getVortices()){
		idx1=this->getVortexIdxUid(n->getUid());
		for(std::weak_ptr<Edge> e : n->getEdges()){
			idx2 = this->getVortexIdxUid(n->getConnectedNode(e.lock())->getUid());
			//std::cout << "this=" << n->getUid() << "   connected=" << n->getConnectedNode(e.lock())->getUid() << std::endl;
			idx = idx1*this->Lattice::getVortices().size() + idx2;
			//std::cout << "idx1=" << idx1 << "   idx2=" << idx2 << " idx=" << idx << "\n" << std::endl;
			mat[idx] = this->Lattice::getVortexDistance(n, this->getVortexIdx(idx2));
		}
	}
}

/**
 * Outputs adjacency matrix in format for copy/paste into Mathematica.
 */
void Lattice::adjMatMtca(unsigned int *mat){
	unsigned int size = this->Lattice::getVortices().size();
	std::cout << "{";
	for(size_t ii = 0; ii < size; ++ii){
		std::cout << "{";
		for(size_t jj = 0; jj < size; ++jj){
			std::cout << mat[ii*size + jj];
			if(jj<size-1)
				std::cout <<",";
			else
				std::cout << "}";
		}
		if(ii<size-1)
			std::cout <<",";
		std::cout << std::endl;
	}
	std::cout << "}" << std::endl;
}
void Lattice::adjMatMtca(double *mat){
	unsigned int size = this->Lattice::getVortices().size();
	std::cout << "{";
	for(size_t ii = 0; ii < size; ++ii){
		std::cout << "{";
		for(size_t jj = 0; jj < size; ++jj){
			std::cout << mat[ii*size + jj];
			if(jj<size-1)
				std::cout <<",";
			else
				std::cout << "}";
		}
		if(ii<size-1)
			std::cout <<",";
		std::cout << std::endl;
	}
	std::cout << "}" << std::endl;
}

// Check connection

std::weak_ptr<Edge> Lattice::isConnected(std::shared_ptr<Node> n1, std::shared_ptr<Node> n2){

	if(n1->getUid() != n2->getUid()){
		for(std::weak_ptr<Edge> e1 : n1->getEdges()){
			if(e1.lock()->isMember(n2)){
				return e1;
			}
		}
	}
	return std::weak_ptr<Edge> ();
}

// Swapping indices and ids

void Lattice::swapIdxUid(unsigned int uid1, unsigned int uid2) {
	Lattice::swapIdx(this->getVortexIdxUid(uid1),this->getVortexIdxUid(uid2));
}
void Lattice::swapIdx(unsigned int idx1, unsigned int idx2) {
	std::swap(this->getVortices().at(idx1),this->getVortices().at(idx2));
}
